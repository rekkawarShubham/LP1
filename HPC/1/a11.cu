
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;

__global__ void maxi(int *a,int *b,int n)
{
	int block=256*blockIdx.x;
	int max=0;
	for(int i=block;i<min(256+block,n);i++)
	{
		if(max<a[i])
		{

			max=a[i];
	
		}
	}
	b[blockIdx.x]=max;
}

int main()
{
	cout<<"Enter the size of array: ";
	int n;
	cin>>n;
	int a[n];

	hipEvent_t start,end,start1,end1;

	for(int i=0;i<n;i++)
	{
		a[i]=rand()%n;
	}
	for(int i=0;i<n;i++)
	{
		printf("%d\t",a[i]);
	}
	hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventRecord(start1);
	int max=0;
	for(int i=0;i<n;i++)
	{
		if(a[i]>max)
		{
			max=a[i];
		}
	}
	hipEventRecord(end1);
	hipEventSynchronize(end1);
	float time1=0;
	hipEventElapsedTime(&time1,start1,end1);
	cout<<"\nSequential Processing:";
	cout<<"\nMax="<<max;
	cout<<"\nSequential time="<<time1;

	int *ad,*bd;
	int size=n*sizeof(int);
	
	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);

	int grids=ceil(n*1.0f/256.0f);
	hipMalloc(&bd,grids*sizeof(int));

	dim3 grid(grids,1);
	dim3 block(1,1);

	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	while(n>1)
	{
		maxi<<<grids,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);
	}

	hipEventRecord(end);
	hipEventSynchronize(end);

	float time=0;
	hipEventElapsedTime(&time,start,end);
	
	int ans[2];
	hipMemcpy(ans,ad,4,hipMemcpyDeviceToHost);	
	cout<<"\nParallel Processing:\nMax="<<ans[0]<<endl;
	cout<<"Parallel Time=";
	cout<<time<<endl;
} 
