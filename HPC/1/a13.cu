
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>
using namespace std;
__global__ void sum(int *a,int *b,int n)
{
	int block=256*blockIdx.x;
	int sum=0;
	for(int i=block;i<min(block+256,n);i++)
	{
		sum=sum+a[i];
	}
	b[blockIdx.x]=sum;
}
int main()
{
	cout<<"Enter the no of elements:";
	int n;
	cin>>n;
	int a[n];

	for(int i=0;i<n;i++)
	{
		a[i]=rand()%n;
		cout<<a[i]<<"\t";
	}

	hipEvent_t start1,end1;
	hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventRecord(start1);
	int sum1=0;
	for(int i=0;i<n;i++)
	{
		sum1+=a[i];
	}
	float mean1=0.0f;
	mean1=sum1/(n*1.0f);
	hipEventRecord(end1);
	hipEventSynchronize(end1);
	float time1=0;
	hipEventElapsedTime(&time1,start1,end1);
	cout<<"\nSequential Processing:";
	cout<<"\nSum="<<sum1;
	cout<<"\nMean="<<mean1;
	cout<<"\nSequential time="<<time1;

	int *ad,*bd;
	int size=n*sizeof(int);
	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);

	int grids=ceil(n*1.0f/256.0f);
	hipMalloc(&bd,grids*sizeof(int));
	dim3 grid(grids,1);
	dim3 block(1,1);
	int p=n;
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
	while(n>1)
	{
		sum<<<grid,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);
	}
	hipEventRecord(end);
	hipEventSynchronize(end);
	float time=0;
	hipEventElapsedTime(&time,start,end);
	int add[2];
	n=p;
	hipMemcpy(add,ad,4,hipMemcpyDeviceToHost);
	cout<<"\nSum="<<add[0]<<endl;
	float mean=0.0f;
	mean=add[0]/(n*1.0f);
	cout<<"Mean="<<mean<<endl;
	cout<<"Parallel time="<<time<<endl;
}
