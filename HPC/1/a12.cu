
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;



__global__ void mini1(int *a,int *b,int n)
{
	int block=256*blockIdx.x;
	int mini=7888888;
	for(int i=block;i<min(256+block,n);i++)
	{
		if(mini>a[i])
		{
			mini=a[i];
		}
	}
	b[blockIdx.x]=mini;
}




int main()
{
	cout<<"Enter the size of array:";
	int n;
	cin>>n;
	int a[n];

	hipEvent_t start,end,start1,end1;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventCreate(&start1);
	hipEventCreate(&end1);

	for(int i=0;i<n;i++)
	{
		a[i]=rand()%n;
		cout<<a[i]<<"\t";
	}
	hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventRecord(start1);
	int max=0;
	for(int i=0;i<n;i++)
	{
		if(a[i]>max)
		{
			max=a[i];
		}
	}
	hipEventRecord(end1);
	hipEventSynchronize(end1);
	float time1=0;
	hipEventElapsedTime(&time1,start1,end1);
	cout<<"\nSequential Processing:";
	cout<<"\nMax="<<max;
	cout<<"\nSequential time="<<time1;

	int *ad,*bd;
	int size=n*sizeof(int);
	
	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
	int grids=ceil(n*1.0f/256.0f);
	hipMalloc(&bd,grids*sizeof(int));
	dim3 grid(grids,1);
	dim3 block(1,1);

	hipEventRecord(start);

	while(n>1)

	{

		mini1<<<grids,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);


	}
	
	hipEventRecord(end);
	hipEventSynchronize(end);

	float time=0;
	hipEventElapsedTime(&time,start,end);
	
	int ans[2];

	hipMemcpy(ans,ad,4,hipMemcpyDeviceToHost);
	
	cout<<"\nParallel Processing:\nMin="<<ans[0]<<endl;
	cout<<"Parallel time=";
	cout<<time<<endl;
} 
