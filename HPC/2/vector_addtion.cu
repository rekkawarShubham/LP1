
#include <hip/hip_runtime.h>
#include<math.h>
#include<time.h>
#include<stdexcept>
#include<iostream>

using namespace std;



__global__ void kernel_sum( int* A,  int* B, int* C, int NUMBERofELEMENTS);

void sum( int* A,  int* B, int* C, int n_el);

int main()
{

	int NUMBER_OF_ELEMENTS;
	cout<<"\nEnter number of elements:";
	cin>>NUMBER_OF_ELEMENTS;
	int SIZE  = NUMBER_OF_ELEMENTS*sizeof(int);

	int* hostA = (int*)malloc(SIZE);
	int* hostB = (int*)malloc(SIZE);
	int* hostC = (int*)malloc(SIZE);
	int* ans = (int*)malloc(SIZE);
	int* deviceA,*deviceB,*deviceC;
	hipEvent_t start,end,start1,end1;

	hipEventCreate(&start1);
	hipEventCreate(&end1);
	srand(time(0));
	int i;
	for(i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		hostA[i] = rand()%NUMBER_OF_ELEMENTS;
		hostB[i] = rand()%NUMBER_OF_ELEMENTS;
	}

	hipEventRecord(start1);
	for(i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		ans[i]=hostA[i]+hostB[i];
	}
	hipEventRecord(end1);
	hipEventSynchronize(end1);
	float t1=0;
	hipEventElapsedTime(&t1,start1,end1);


	hipEventCreate(&start);
	hipEventCreate(&end);
	hipMalloc(&deviceA,SIZE);
	hipMalloc(&deviceB,SIZE);
	hipMalloc(&deviceC,SIZE);

	hipMemcpy(deviceA,hostA,SIZE,hipMemcpyHostToDevice);
	hipMemcpy(deviceB,hostB,SIZE,hipMemcpyHostToDevice);

	hipEventRecord(start);

	sum(deviceA,deviceB,deviceC,NUMBER_OF_ELEMENTS);

	hipEventRecord(end);
	hipEventSynchronize(end);
	float t=0;
	hipEventElapsedTime(&t,start,end);
    hipMemcpy(hostC,deviceC,SIZE,hipMemcpyDeviceToHost);


	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	double error = 0;
	for(i = 0;i<NUMBER_OF_ELEMENTS;i++)
	{
		double diff = double((hostA[i]+hostB[i])-hostC[i]);
		error+=diff;
		cout<<"\nExpected value="<<ans[i];
		cout<<"\tActual value="<<hostC[i];
	}

	error = sqrt(error);
	cout<<"\nError  = "<<error<<endl;
	cout<<"\nSequential time="<<t1;
	cout<<"\nParallel time="<<t<<endl;	
	delete[] hostA;
    delete[] hostB;
    delete[] hostC;
    return hipDeviceSynchronize();
}


void sum( int* A,  int* B, int* C, int n_el)
{
	int threadsPerblock,blocksperGrid;

	if(n_el<512)
	{
		threadsPerblock = n_el;
		blocksperGrid = 1;
	}
	else
	{
		threadsPerblock = 512;
		blocksperGrid = ceil(double(n_el)/double(threadsPerblock));
	}

	//now invoke kernel method
	kernel_sum<<<blocksperGrid,threadsPerblock>>>(A,B,C,n_el);
}


__global__ void kernel_sum( int* A,  int* B, int* C, int NUMBERofELEMENTS)
{
	//calculate unique thread index

	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if(index<NUMBERofELEMENTS)
	C[index] = A[index] + B[index];
}
