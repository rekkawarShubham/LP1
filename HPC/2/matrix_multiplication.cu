
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#include<cstdlib>
#include<stdlib.h>

using namespace std;
__global__ void matrixMultiplication(int* A,int* B,int* C,int N);


void mm(int* A,int* B,int* C,int N);

int main()
{
	hipEvent_t start,end,start1,end1;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventCreate(&start1);
	hipEventCreate(&end1);

	int ROWS = 1<<2;
	int COLS = 1<<2;

	cout<<"\nEnter number of rows:";
	cin>>ROWS;

	cout<<"\nEnter number of cols:";
	cin>>COLS;

	int* hostA = (int*)malloc(sizeof(int)*ROWS*COLS);
	int* hostB = (int*)malloc(sizeof(int)*ROWS*COLS);
	int* hostC = (int*)malloc(sizeof(int)*ROWS*COLS);
	srand(time(0));
	int i,j;
	for(i=0;i<ROWS;i++)
	{
		for(j=0;j<COLS;j++)
		{
			hostB[i*COLS+j] = rand()%30;
			hostA[i*COLS+j] = rand()%20;
		}
	}
	cout<<"\nMatrix A:\n";
	for(i=0;i<ROWS;i++)
	{
		for(j=0;j<COLS;j++)
		{
			//cout<<hostA[i*COLS+j]<<"\t";
		}
		//cout<<"\n";
	}

	cout<<"\nMatrix B:\n";
	for(i=0;i<ROWS;i++)
	{
		for(j=0;j<COLS;j++)
		{
			//cout<<hostB[i*COLS+j]<<"\t";
		}
		//cout<<"\n";
	}


	int* deviceA,*deviceB,*deviceC;
	hipMalloc(&deviceA,sizeof(int)*ROWS*COLS);
	hipMalloc(&deviceB,sizeof(int)*ROWS*COLS);
	hipMalloc(&deviceC,sizeof(int)*ROWS*COLS);
	hipMemcpy(deviceA,hostA,sizeof(int)*ROWS*COLS,hipMemcpyHostToDevice);
	hipMemcpy(deviceB,hostB,sizeof(int)*ROWS*COLS,hipMemcpyHostToDevice);

	hipEventRecord(start);

	mm(deviceA,deviceB,deviceC,ROWS);

	hipEventRecord(end);
	hipEventSynchronize(end);
	float t=0;
	hipEventElapsedTime(&t,start,end);

	hipError_t e=hipGetLastError();                                 
 	if(e!=hipSuccess) 
 	{                                              
   		printf("Cuda failure %s: ",hipGetErrorString(e));           
   	}     
	hipDeviceSynchronize();
    hipMemcpy(hostC,deviceC,ROWS*COLS*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	hipEventRecord(start1);
	int N = ROWS;
	int* actual = (int*)malloc(sizeof(int)*ROWS*COLS);
	int sum;
	for (int row=0; row<ROWS; row++)
	{
        for (int col=0; col<COLS; col++)
        {
            sum=0;
            for (int n=0; n<N; n++)
            {
                sum += hostA[row*N+n]*hostB[n*N+col];
            }
            actual[row*N+col] = sum;
        }
    }
    hipEventRecord(end1);
	hipEventSynchronize(end1);
	float t1=0;
	hipEventElapsedTime(&t1,start1,end1);

    double error = 0;
    for(int k=0;k<ROWS*COLS;k++)
	{
		cout<<k<<")"<< "Expected value = "<<actual[k]<<"\tActual value = "<<hostC[k]<<"\n";
		error += double(abs(actual[k]-hostC[k]));
	}
	error=sqrt(error);
	cout<<"error = "<<error<<"\n";
	delete[] hostA;
    delete[] hostB;
    delete[] hostC;
    cout<<"\nSequential time="<<t1;
    cout<<"\nParallel time="<<t<<endl;
}

__global__ void matrixMultiplication(int* A,int* B,int* C,int N)
{
	int ROW = blockIdx.y*blockDim.y+threadIdx.y;
	int COL = blockIdx.x*blockDim.x+threadIdx.x;
	int sum =0 ;
	if(ROW<N && COL<N)
	{
		for(int i=0;i<N;i++)
		{
			sum+=A[ROW*N+i]*B[i*N+COL];
		}
		__syncthreads();    
		C[ROW*N+COL]=sum;
	}
	
}


void mm(int* A,int* B,int* C,int N)
{
	dim3 threadsPerblock(N,N);
	dim3 blocksPerGrid(1,1);
	if(N*N>512)
	{
		threadsPerblock.x = 512;
		threadsPerblock.y=512;
		blocksPerGrid.x = ceil(double(N)/double(threadsPerblock.x));
		blocksPerGrid.y = ceil(double(N)/double(threadsPerblock.y));
	}
	matrixMultiplication<<<blocksPerGrid,threadsPerblock>>>(A,B,C,N);
}