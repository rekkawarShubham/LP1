
#include <hip/hip_runtime.h>
#include<math.h>
#include<time.h>
#include<stdexcept>
#include<iostream>
#include<cstdlib> //for abs(x)
#include<stdio.h>

using namespace std;



__global__ void kernel_multiplication( int* A,  int* B, int* C,int N,int M);


int main()
{
	int NUMBER_OF_ELEMENTS;
	int VECTOR_SIZE;
	cout<<"Enter the vector size:";
	cin>>VECTOR_SIZE;
	NUMBER_OF_ELEMENTS=VECTOR_SIZE;
	int SIZE  = NUMBER_OF_ELEMENTS*sizeof(int);


	hipEvent_t start,end,start1,end1;

	int* hostA = (int*)malloc(VECTOR_SIZE*sizeof(int));
	int* hostB = (int*)malloc(SIZE*VECTOR_SIZE*sizeof(int));
	int* hostC = (int*)malloc(VECTOR_SIZE*sizeof(int));

	int* deviceA,*deviceB,*deviceC;

	srand(time(0));
	int i,j;

	cout<<"\nVector:\n";
	for(i=0;i<VECTOR_SIZE;i++)
	{
		hostA[i] = rand()%VECTOR_SIZE;	
		cout<<hostA[i]<<"\t";
	}

	//initialize matrix by random elements
	for(i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		for(j=0;j<VECTOR_SIZE;j++)
		{
			hostB[i*VECTOR_SIZE+j] = rand()%VECTOR_SIZE;
		}
	}

	cout<<"\nMatrix=\n";
	for(i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		for(j=0;j<VECTOR_SIZE;j++)
		{
			cout<<hostB[i*VECTOR_SIZE+j]<<"\t";
		}
		cout<<"\n";
	}

	hipMalloc(&deviceA,VECTOR_SIZE*sizeof(int));
	hipMalloc(&deviceB,NUMBER_OF_ELEMENTS*VECTOR_SIZE*sizeof(int));
	hipMalloc(&deviceC,VECTOR_SIZE*sizeof(int));

	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventCreate(&start1);
	hipEventCreate(&end1);

	hipEventRecord(start);
	hipMemcpy(deviceA,hostA,VECTOR_SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(deviceB,hostB,SIZE*VECTOR_SIZE,hipMemcpyHostToDevice);
	kernel_multiplication<<<NUMBER_OF_ELEMENTS,1>>>(deviceA,deviceB,deviceC,NUMBER_OF_ELEMENTS,VECTOR_SIZE);
	hipDeviceSynchronize();
    hipMemcpy(hostC,deviceC,VECTOR_SIZE*sizeof(int),hipMemcpyDeviceToHost);
    hipEventRecord(end);
    hipEventSynchronize(end);
	float t=0;
	hipEventElapsedTime(&t,start,end);


	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	double error = 0;

    int* answer = (int*) malloc(VECTOR_SIZE*sizeof(int));

    hipEventRecord(start1);
	for(int i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		int sum = 0;
		for(int j=0;j<VECTOR_SIZE;j++)
		{
			sum += hostA[j]*hostB[j*VECTOR_SIZE+i];
		}
		answer[i] = sum;
	}

	for(int k=0;k<VECTOR_SIZE;k++)
	{
		cout<<k<<")"<< "Expected value = "<<answer[k]<<" Actual value = "<<hostC[k]<<"\n";
		error += double(abs(answer[k]-hostC[k]));
	}

	error=sqrt(error);
	cout<<"error = "<<error<<"\n";
	hipEventRecord(end1);
    hipEventSynchronize(end1);
	float t1=0;
	hipEventElapsedTime(&t1,start1,end1);

	cout<<"\nSequential time="<<t1;
	cout<<"\nParallel time="<<t<<endl;

	delete[] hostA;
    delete[] hostB;
    delete[] hostC;
    return hipDeviceSynchronize();

}

__global__ void kernel_multiplication( int* A,  int* B, int* C, int N,int M)
{
	int index =  threadIdx.x + blockIdx.x * blockDim.x;
	int sum = 0;
	if(index<N)
	{
		for(int i=0;i<M;i++)
		sum+=A[i]*B[(i*M)+index];
		C[index] = sum;
	}
}